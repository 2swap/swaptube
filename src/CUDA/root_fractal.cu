#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <cmath>
#include <cstdio>

__device__ hipFloatComplex complex_pow(hipFloatComplex z, int n) {
    hipFloatComplex result = make_hipFloatComplex(1.0f, 0.0f);
    for (int i = 0; i < n; i++) {
        result = hipCmulf(result, z);
    }
    return result;
}

// Evaluate polynomial p(z) = coeffs[0] + coeffs[1] z + ... + coeffs[n] z^n
__device__ hipFloatComplex eval_poly(const hipFloatComplex* coeffs, int n, hipFloatComplex z) {
    hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
    for (int i = 0; i <= n; i++) {
        sum = hipCaddf(sum, hipCmulf(coeffs[i], complex_pow(z, i)));
    }
    return sum;
}

// Evaluate derivative of polynomial
__device__ hipFloatComplex eval_poly_derivative(const hipFloatComplex* coeffs, int n, hipFloatComplex z) {
    hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
    for (int i = 1; i <= n; i++) {
        sum = hipCaddf(sum, make_hipFloatComplex(i * hipCrealf(coeffs[i]), i * hipCimagf(coeffs[i])));
        sum = hipCaddf(sum, hipCmulf(make_hipFloatComplex(i, 0.0f), complex_pow(z, i-1)));
    }
    return sum;
}

// Simple Newton-Raphson root finder for complex polynomials
__device__ hipFloatComplex find_root(const hipFloatComplex* coeffs, int n, hipFloatComplex z0, int max_iter=50, float tol=1e-4f) {
    hipFloatComplex z = z0;
    for (int i = 0; i < max_iter; i++) {
        hipFloatComplex f = eval_poly(coeffs, n, z);
        hipFloatComplex fprime = eval_poly_derivative(coeffs, n, z);
        float fprime_mag2 = hipCrealf(fprime)*hipCrealf(fprime) + hipCimagf(fprime)*hipCimagf(fprime);
        if (fprime_mag2 < 1e-8f) break; // avoid division by zero
        hipFloatComplex dz = hipCdivf(f, fprime);
        z = hipCsubf(z, dz);
        if (hipCabsf(dz) < tol) break;
    }
    return z;
}

__global__ void root_fractal_kernel(int* pixels, int w, int h, hipFloatComplex c1, hipFloatComplex c2, int n) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total = 1 << n; // total number of polynomials

    if (idx >= total) return;

    hipFloatComplex coeffs[20];
    for (int i = 0; i <= n; i++) {
        coeffs[i] = (idx & (1 << i)) ? c2 : c1;
    }

    // try roots starting from points on a circle
    for (int i = 0; i < n; i++) {
        float angle = 2.0f * 3.14159265359f * i / n;
        hipFloatComplex z0 = make_hipFloatComplex(cosf(angle), sinf(angle));
        hipFloatComplex root = find_root(coeffs, n, z0);

        // map root to pixel coordinates
        int px = (int)((root.x + 2.0f) / 4.0f * w);
        int py = (int)((root.y + 2.0f) / 4.0f * h);

        if (px >= 0 && px < w && py >= 0 && py < h) {
            int offset = 3 * (py * w + px);
            pixels[offset] = 0xffffffff;
        }
    }
}

extern "C" void draw_root_fractal(int* pixels, int w, int h, complex<float> c1, complex<float> c2, int n) {
    int total = 1 << n;
    int* d_pixels;
    hipMalloc(&d_pixels, w * h * sizeof(int));
    hipMemcpy(d_pixels, pixels, w * h * sizeof(int), hipMemcpyHostToDevice);

    hipFloatComplex dc1 = make_hipFloatComplex(c1.real(), c1.imag());
    hipFloatComplex dc2 = make_hipFloatComplex(c2.real(), c2.imag());

    int threadsPerBlock = 256;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;

    root_fractal_kernel<<<blocks, threadsPerBlock>>>(d_pixels, w, h, dc1, dc2, n);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, w * h * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}

