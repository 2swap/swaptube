#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <cmath>
#include <cstdio>
#include "cuda_graphics.cu"

__device__ glm::vec2 point_to_pixel(const glm::vec2& point, const glm::vec2& lx_ty, const glm::vec2& rx_by, const glm::vec2& wh) {
    const glm::vec2 flip = (point - lx_ty) * wh / (rx_by - lx_ty);
    return glm::vec2(flip.x, wh.y-1-flip.y);
}

__device__ hipFloatComplex complex_pow(hipFloatComplex z, int n) {
    hipFloatComplex result = make_hipFloatComplex(1.0f, 0.0f);
    for (int i = 0; i < n; i++) {
        result = hipCmulf(result, z);
    }
    return result;
}

__device__ void find_roots(const hipFloatComplex* coeffs_in, int degree, hipFloatComplex* roots) {
    // Use Durand-Kerner (Weierstrass) method to find all roots of polynomial
    // coeffs_in: coeffs[0..degree] where coeffs[i] corresponds to x^i
    // degree: n (degree of polynomial)
    if (degree <= 0) {
        return;
    }

    const int maxn = 20;
    hipFloatComplex coeffs[maxn + 1];
    for (int i = 0; i <= degree; ++i) {
        coeffs[i] = coeffs_in[i];
    }

    // Make monic: divide all coefficients by leading coefficient coeffs[degree]
    hipFloatComplex leading = coeffs[degree];
    float leading_abs = hipCabsf(leading);
    if (leading_abs == 0.0f) {
        // Degenerate polynomial; just return zeros
        for (int i = 0; i < degree; ++i) roots[i] = make_hipFloatComplex(0.0f, 0.0f);
        return;
    }
    for (int i = 0; i <= degree; ++i) {
        coeffs[i] = hipCdivf(coeffs[i], leading);
    }

    // Compute radius for initial guesses: 1 + max |a_i| for i=0..degree-1
    float max_coeff_abs = 0.0f;
    for (int i = 0; i < degree; ++i) {
        float aabs = hipCabsf(coeffs[i]);
        if (aabs > max_coeff_abs) max_coeff_abs = aabs;
    }
    float radius = 1.0f + max_coeff_abs;

    // Initialize roots on a circle
    const float PI2 = 6.28318530717958647692f;
    for (int i = 0; i < degree; ++i) {
        float angle = PI2 * i / degree;
        roots[i] = make_hipFloatComplex(radius * cosf(angle), radius * sinf(angle));
    }

    const int max_iters = 100;
    const float tol = 1e-6f;

    for (int iter = 0; iter < max_iters; ++iter) {
        float max_change = 0.0f;

        // For each root
        for (int i = 0; i < degree; ++i) {
            hipFloatComplex xi = roots[i];

            // Evaluate polynomial p(xi) using Horner's method: coeffs[degree]*x^degree + ... + coeffs[0]
            hipFloatComplex p = coeffs[degree];
            for (int k = degree - 1; k >= 0; --k) {
                p = hipCaddf(hipCmulf(p, xi), coeffs[k]);
            }

            // Compute denominator: product_{j != i} (xi - xj)
            hipFloatComplex denom = make_hipFloatComplex(1.0f, 0.0f);
            for (int j = 0; j < degree; ++j) {
                if (j == i) continue;
                hipFloatComplex diff = hipCsubf(xi, roots[j]);
                float diff_abs = hipCabsf(diff);
                if (diff_abs == 0.0f) {
                    // Perturb slightly to avoid zero division
                    diff = hipCaddf(diff, make_hipFloatComplex(1e-6f, 1e-6f));
                }
                denom = hipCmulf(denom, diff);
            }

            float denom_abs = hipCabsf(denom);
            if (denom_abs == 0.0f) continue;

            hipFloatComplex correction = hipCdivf(p, denom);
            hipFloatComplex new_xi = hipCsubf(xi, correction);
            float change = hipCabsf(hipCsubf(new_xi, xi));
            if (change > max_change) max_change = change;
            roots[i] = new_xi;
        }

        if (max_change < tol) break;
    }
}

__global__ void root_fractal_kernel(unsigned int* pixels, int w, int h, hipFloatComplex c1, hipFloatComplex c2, float terms, float lx, float ty, float rx, float by, float radius, float opacity) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ceil_terms = ceil(terms);
    unsigned int floor_terms = floor(terms);
    unsigned int total = 1 << ceil_terms; // total number of polynomials

    // Multiply coefficients by 100 to avoid numerical issues
    c1 = hipCmulf(c1, make_hipFloatComplex(100.0f, 0.0f));
    c2 = hipCmulf(c2, make_hipFloatComplex(100.0f, 0.0f));

    if (idx >= total) return;
    if (idx >= total/2 && floor_terms != ceil_terms) {
        float one_minus_frac = 1 - (terms - floor_terms);
        float radius_multiplier = 1-one_minus_frac*one_minus_frac*one_minus_frac;
        radius *= radius_multiplier;
    }

    hipFloatComplex coeffs[20];
    unsigned int color = 0xFF3f3f3f;
    for (int i = 0; i < ceil_terms; i++) {
        bool bit = (idx >> i) & 1;
        coeffs[i] = bit ? c2 : c1;
        if(!bit) continue;
        unsigned int color_or = 1 << 23;
        color_or >>= i/3;
        color_or >>= i%3 * 8;
        color |= color_or;
    }

    // Find the degree, since the leading coefficients might be zero
    int degree = -1;
    for (int i = ceil_terms - 1; i >= 0; i--) {
        if (coeffs[i].x != 0.0f || coeffs[i].y != 0.0f) {
            degree = i;
            break;
        }
    }
    if(degree < 1) return;

    hipFloatComplex roots[20];
    find_roots(coeffs, degree, roots);

    // Plot the roots
    for (int i = 0; i < degree; i++) {
        glm::vec2 point(hipCrealf(roots[i]), hipCimagf(roots[i]));
        glm::vec2 pixel = point_to_pixel(point, glm::vec2(lx, ty), glm::vec2(rx, by), glm::vec2(w, h));
        device_gradient_circle(pixel.x, pixel.y, radius, color, pixels, w, h, opacity);
    }
}

extern "C" void draw_root_fractal(
    unsigned int* pixels,
    int w,
    int h,
    complex<float> c1,
    complex<float> c2,
    float terms,
    float lx, float ty,
    float rx, float by,
    float radius, float opacity
) {
    int total = 1 << int(ceil(terms));
    unsigned int* d_pixels;
    hipMalloc(&d_pixels, w * h * sizeof(unsigned int));
    hipMemcpy(d_pixels, pixels, w * h * sizeof(unsigned int), hipMemcpyHostToDevice);

    hipFloatComplex dc1 = make_hipFloatComplex(c1.real(), c1.imag());
    hipFloatComplex dc2 = make_hipFloatComplex(c2.real(), c2.imag());

    int threadsPerBlock = 256;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;

    root_fractal_kernel<<<blocks, threadsPerBlock>>>(d_pixels, w, h, dc1, dc2, terms, lx, ty, rx, by, radius, opacity);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, w * h * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}
