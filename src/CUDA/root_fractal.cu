#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <cmath>
#include <cstdio>
#include "cuda_graphics.cu"

__device__ glm::vec2 point_to_pixel(const glm::vec2& point, const glm::vec2& lx_ty, const glm::vec2& rx_by, const glm::vec2& wh) {
    const glm::vec2 flip = (point - lx_ty) * wh / (rx_by - lx_ty);
    return glm::vec2(flip.x, wh.y-1-flip.y);
}

__device__ hipFloatComplex complex_pow(hipFloatComplex z, int n) {
    hipFloatComplex result = make_hipFloatComplex(1.0f, 0.0f);
    for (int i = 0; i < n; i++) {
        result = hipCmulf(result, z);
    }
    return result;
}

__device__ void find_roots(const hipFloatComplex* coeffs_in, int degree, hipFloatComplex* roots) {
    // Use Durand-Kerner (Weierstrass) method to find all roots of polynomial
    // coeffs_in: coeffs[0..degree] where coeffs[i] corresponds to x^i
    // degree: n (degree of polynomial)
    if (degree <= 0) {
        return;
    }

    const int maxn = 20;
    hipFloatComplex coeffs[maxn + 1];
    for (int i = 0; i <= degree; ++i) {
        coeffs[i] = coeffs_in[i];
    }

    // Make monic: divide all coefficients by leading coefficient coeffs[degree]
    hipFloatComplex leading = coeffs[degree];
    float leading_abs = hipCabsf(leading);
    if (leading_abs == 0.0f) {
        // Degenerate polynomial; just return zeros
        for (int i = 0; i < degree; ++i) roots[i] = make_hipFloatComplex(0.0f, 0.0f);
        return;
    }
    for (int i = 0; i <= degree; ++i) {
        coeffs[i] = hipCdivf(coeffs[i], leading);
    }

    // Compute radius for initial guesses: 1 + max |a_i| for i=0..degree-1
    float max_coeff_abs = 0.0f;
    for (int i = 0; i < degree; ++i) {
        float aabs = hipCabsf(coeffs[i]);
        if (aabs > max_coeff_abs) max_coeff_abs = aabs;
    }
    float radius = 1.0f + max_coeff_abs;

    // Initialize roots on a circle
    const float PI2 = 6.28318530717958647692f;
    for (int i = 0; i < degree; ++i) {
        float angle = PI2 * i / degree;
        roots[i] = make_hipFloatComplex(radius * cosf(angle), radius * sinf(angle));
    }

    const int max_iters = 200;
    const float tol = 1e-6f;

    for (int iter = 0; iter < max_iters; ++iter) {
        float max_change = 0.0f;

        // For each root
        for (int i = 0; i < degree; ++i) {
            hipFloatComplex xi = roots[i];

            // Evaluate polynomial p(xi) using Horner's method: coeffs[degree]*x^degree + ... + coeffs[0]
            hipFloatComplex p = coeffs[degree];
            for (int k = degree - 1; k >= 0; --k) {
                p = hipCaddf(hipCmulf(p, xi), coeffs[k]);
            }

            // Compute denominator: product_{j != i} (xi - xj)
            hipFloatComplex denom = make_hipFloatComplex(1.0f, 0.0f);
            for (int j = 0; j < degree; ++j) {
                if (j == i) continue;
                hipFloatComplex diff = hipCsubf(xi, roots[j]);
                float diff_abs = hipCabsf(diff);
                if (diff_abs == 0.0f) {
                    // Perturb slightly to avoid zero division
                    diff = hipCaddf(diff, make_hipFloatComplex(1e-6f, 1e-6f));
                }
                denom = hipCmulf(denom, diff);
            }

            float denom_abs = hipCabsf(denom);
            if (denom_abs == 0.0f) continue;

            hipFloatComplex correction = hipCdivf(p, denom);
            hipFloatComplex new_xi = hipCsubf(xi, correction);
            float change = hipCabsf(hipCsubf(new_xi, xi));
            if (change > max_change) max_change = change;
            roots[i] = new_xi;
        }

        if (max_change < tol) break;
    }
}

__global__ void root_fractal_kernel(unsigned int* pixels, int w, int h, hipFloatComplex c1, hipFloatComplex c2, int terms, float lx, float ty, float rx, float by) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total = 1 << terms; // total number of polynomials
    int degree = terms - 1;

    if (idx >= total) return;

    hipFloatComplex coeffs[20];
    unsigned int color = 0xFF000000;
    for (int i = 0; i < terms; i++) {
        bool bit = (idx >> i) & 1;
        coeffs[i] = bit ? c2 : c1;
        if(!bit) continue;
        unsigned int color_or = 1 << 23;
        color_or >>= i/3;
        color_or >>= i%3 * 8;
        color |= color_or;
    }

    hipFloatComplex roots[20];
    find_roots(coeffs, degree, roots);

    // Plot the roots
    for (int i = 0; i < degree; i++) {
        glm::vec2 point(hipCrealf(roots[i]), hipCimagf(roots[i]));
        glm::vec2 pixel = point_to_pixel(point, glm::vec2(lx, ty), glm::vec2(rx, by), glm::vec2(w, h));
        int px = static_cast<int>(roundf(pixel.x));
        int py = static_cast<int>(roundf(pixel.y));
        if (px >= 0 && px < w && py >= 0 && py < h) {
            device_fill_circle(px, py, 1, color, pixels, w, h);
        }
    }
}

extern "C" void draw_root_fractal(
    unsigned int* pixels,
    int w,
    int h,
    complex<float> c1,
    complex<float> c2,
    int terms,
    float lx, float ty,
    float rx, float by
) {
    int total = 1 << terms;
    unsigned int* d_pixels;
    hipMalloc(&d_pixels, w * h * sizeof(unsigned int));
    hipMemcpy(d_pixels, pixels, w * h * sizeof(unsigned int), hipMemcpyHostToDevice);

    hipFloatComplex dc1 = make_hipFloatComplex(c1.real(), c1.imag());
    hipFloatComplex dc2 = make_hipFloatComplex(c2.real(), c2.imag());

    int threadsPerBlock = 256;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;

    root_fractal_kernel<<<blocks, threadsPerBlock>>>(d_pixels, w, h, dc1, dc2, terms, lx, ty, rx, by);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, w * h * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}
