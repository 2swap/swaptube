#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>

#define GRID_SIZE 10 // 10x10x10 bins
#define BIN_INDEX(x, y, z) ((z) * GRID_SIZE * GRID_SIZE + (y) * GRID_SIZE + (x))

struct Bin {
    int count;
    glm::dvec4 center_of_mass;
};

__device__ glm::dvec4 compute_force(glm::dvec4 pos_i, glm::dvec4 pos_j) {
    glm::dvec4 diff = pos_i - pos_j;
    double dist_sq = glm::dot(diff, diff) + 50; // Add epsilon to avoid division by zero
    return glm::normalize(diff) / dist_sq;
}

__global__ void compute_repulsion_kernel_naive(const glm::dvec4* positions, glm::dvec4* velocity_deltas,
                                               int num_nodes, double repel_force) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_nodes) return;

    glm::dvec4 pos_i = positions[i];
    glm::dvec4 delta = glm::dvec4(0.0);

    for (int j = 0; j < num_nodes; ++j) {
        if (i == j) continue;

        delta += compute_force(pos_i, positions[j]);
    }

    velocity_deltas[i] = repel_force * delta;
}


void sort_positions_by_bins_with_indices(const glm::dvec4* positions, glm::dvec4* sorted_positions, 
                                         const int* node_bins, int* sorted_node_bins, 
                                         int* original_indices, int* sorted_indices,
                                         int num_nodes, int* bin_counts) {
    // Step 1: Compute cumulative bin counts (prefix sum) to determine sorted indices
    int num_bins = GRID_SIZE * GRID_SIZE * GRID_SIZE;
    int* bin_offsets = new int[num_bins + 1];
    bin_offsets[0] = 0;

    for (int i = 0; i < num_bins; ++i) {
        bin_offsets[i + 1] = bin_offsets[i] + bin_counts[i];
    }

    // Step 2: Insert nodes into their sorted positions
    for (int i = 0; i < num_nodes; ++i) {
        int bin_idx = node_bins[i];
        int sorted_idx = bin_offsets[bin_idx]++; // Get the current position and increment the offset

        sorted_positions[sorted_idx] = positions[i];
        sorted_node_bins[sorted_idx] = bin_idx;
        sorted_indices[sorted_idx] = original_indices[i];
    }

    // Cleanup
    delete[] bin_offsets;
}

__global__ void compute_repulsion_kernel_binned(const glm::dvec4* sorted_positions, glm::dvec4* velocity_deltas,
                                                const Bin* bins, const int* bin_start_indices, 
                                                const int* sorted_indices, int num_nodes, 
                                                double repel_force, glm::dvec4 min_bounds, glm::dvec4 bin_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_nodes) return;

    glm::dvec4 pos_i = sorted_positions[i];
    glm::dvec4 delta = glm::dvec4(0.0);

    // Determine the bin index for the current node
    glm::ivec3 bin_idx = glm::ivec3((pos_i - min_bounds) / bin_size);
    bin_idx = glm::clamp(bin_idx, glm::ivec3(0), glm::ivec3(GRID_SIZE - 1));

    // **Step 1: Interact with nodes in neighboring bins**
    for (int dz = -1; dz <= 1; ++dz) {
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                glm::ivec3 neighbor_bin_idx = bin_idx + glm::ivec3(dx, dy, dz);

                // Skip bins outside valid range
                if (neighbor_bin_idx.x < 0 || neighbor_bin_idx.x >= GRID_SIZE ||
                    neighbor_bin_idx.y < 0 || neighbor_bin_idx.y >= GRID_SIZE ||
                    neighbor_bin_idx.z < 0 || neighbor_bin_idx.z >= GRID_SIZE) {
                    continue;
                }

                int neighbor_bin_flat_idx = BIN_INDEX(neighbor_bin_idx.x, neighbor_bin_idx.y, neighbor_bin_idx.z);
                int start_idx = bin_start_indices[neighbor_bin_flat_idx];
                int end_idx = bin_start_indices[neighbor_bin_flat_idx + 1];

                for (int j = start_idx; j < end_idx; ++j) {
                    if (i == j) continue; // Skip self-interaction

                    delta += compute_force(pos_i, sorted_positions[j]);
                }
            }
        }
    }

    // **Step 2: Interact with non-neighboring bins (bin-level interaction)**
    for (int z = 0; z < GRID_SIZE; ++z) {
        for (int y = 0; y < GRID_SIZE; ++y) {
            for (int x = 0; x < GRID_SIZE; ++x) {
                glm::ivec3 current_bin_idx_3d(x, y, z);

                // Skip the 3x3x3 neighborhood
                if (abs(current_bin_idx_3d.x - bin_idx.x) <= 1 &&
                    abs(current_bin_idx_3d.y - bin_idx.y) <= 1 &&
                    abs(current_bin_idx_3d.z - bin_idx.z) <= 1) {
                    continue;
                }

                int non_neighbor_bin_idx = BIN_INDEX(current_bin_idx_3d.x, current_bin_idx_3d.y, current_bin_idx_3d.z);
                Bin bin = bins[non_neighbor_bin_idx];

                if (bin.count == 0) continue; // Skip empty bins

                delta += (double)bin.count * compute_force(pos_i, bin.center_of_mass);
            }
        }
    }

    // Map back to the original index
    int original_idx = sorted_indices[i];
    velocity_deltas[original_idx] = repel_force * delta;
}

__device__ double atomicMin_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        double old_val = __longlong_as_double(assumed);
        if (old_val <= val) break; // If the current value is already smaller, no need to update
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicMax_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        double old_val = __longlong_as_double(assumed);
        if (old_val >= val) break; // If the current value is already larger, no need to update
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(__longlong_as_double(assumed) + val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void populate_bins(const glm::dvec4* positions, Bin* bins, int num_nodes, glm::dvec4 min_bounds, glm::dvec4 bin_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_nodes) return;

    glm::dvec4 pos = positions[i];
    glm::ivec3 bin_idx = glm::ivec3((pos - min_bounds) / bin_size);
    bin_idx = glm::clamp(bin_idx, glm::ivec3(0), glm::ivec3(GRID_SIZE - 1));

    int bin_flat_idx = BIN_INDEX(bin_idx.x, bin_idx.y, bin_idx.z);

    // Atomic operations to update the bin's data
    atomicAdd(&bins[bin_flat_idx].count, 1);
    atomicAdd_double(&bins[bin_flat_idx].center_of_mass.x, pos.x);
    atomicAdd_double(&bins[bin_flat_idx].center_of_mass.y, pos.y);
    atomicAdd_double(&bins[bin_flat_idx].center_of_mass.z, pos.z);
    atomicAdd_double(&bins[bin_flat_idx].center_of_mass.w, pos.w);
}

__global__ void finalize_bins(Bin* bins, int num_bins) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_bins) return;

    if (bins[i].count > 0) {
        bins[i].center_of_mass /= double(bins[i].count);
    }
}

struct AABB {
    glm::dvec4 min_bounds;
    glm::dvec4 max_bounds;
};

__global__ void compute_aabb(const glm::dvec4* positions, int num_nodes, glm::dvec4* min_bounds, glm::dvec4* max_bounds) {
    __shared__ glm::dvec4 local_min;
    __shared__ glm::dvec4 local_max;

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory
    if (tid == 0) {
        local_min = glm::dvec4(DBL_MAX);
        local_max = glm::dvec4(-DBL_MAX);
    }
    __syncthreads();

    // Update local min/max
    if (i < num_nodes) {
        atomicMin_double(&local_min.x, positions[i].x);
        atomicMin_double(&local_min.y, positions[i].y);
        atomicMin_double(&local_min.z, positions[i].z);

        atomicMax_double(&local_max.x, positions[i].x);
        atomicMax_double(&local_max.y, positions[i].y);
        atomicMax_double(&local_max.z, positions[i].z);
    }
    __syncthreads();

    // Update global min/max using shared memory
    if (tid == 0) {
        atomicMin_double(&min_bounds->x, local_min.x);
        atomicMin_double(&min_bounds->y, local_min.y);
        atomicMin_double(&min_bounds->z, local_min.z);

        atomicMax_double(&max_bounds->x, local_max.x);
        atomicMax_double(&max_bounds->y, local_max.y);
        atomicMax_double(&max_bounds->z, local_max.z);
    }
}

void compute_node_bins(const glm::dvec4* positions, int* node_bins, int num_nodes, glm::dvec4 min_bounds, glm::dvec4 bin_size) {
    for (int i = 0; i < num_nodes; ++i) {
        glm::ivec3 bin_idx = glm::ivec3((positions[i] - min_bounds) / bin_size);
        bin_idx = glm::clamp(bin_idx, glm::ivec3(0), glm::ivec3(GRID_SIZE - 1));
        node_bins[i] = BIN_INDEX(bin_idx.x, bin_idx.y, bin_idx.z);
    }
}

extern "C" void compute_repulsion_cuda(const glm::dvec4* host_positions, glm::dvec4* host_velocity_deltas, 
                                       int num_nodes, double repel_force) {
    glm::dvec4 *d_positions, *d_velocity_deltas;

    size_t size = num_nodes * sizeof(glm::dvec4);

    // Allocate device memory
    hipMalloc(&d_positions, size);
    hipMalloc(&d_velocity_deltas, size);

    // Copy positions to device
    hipMemcpy(d_positions, host_positions, size, hipMemcpyHostToDevice);
    hipMemset(d_velocity_deltas, 0, size);

    int blockSize = 128;
    int gridSize = (num_nodes + blockSize - 1) / blockSize;

    if (num_nodes < 5000) {
        // Use naive algorithm for small graphs
        compute_repulsion_kernel_naive<<<gridSize, blockSize>>>(d_positions, d_velocity_deltas, num_nodes, repel_force);
    } else {
        // Use binned algorithm for larger graphs

        // Host data for bounds and bin size
        glm::dvec4 h_min_bounds(DBL_MAX, DBL_MAX, DBL_MAX, DBL_MAX);
        glm::dvec4 h_max_bounds(-DBL_MAX, -DBL_MAX, -DBL_MAX, -DBL_MAX);
        glm::dvec4 h_bin_size;

        Bin* d_bins;
        int* d_node_bins;

        size_t bin_size = GRID_SIZE * GRID_SIZE * GRID_SIZE * sizeof(Bin);
        size_t node_bins_size = num_nodes * sizeof(int);

        hipMalloc(&d_bins, bin_size);
        hipMalloc(&d_node_bins, node_bins_size);

        hipMemset(d_bins, 0, bin_size);

        // Step 1: Compute AABB
        glm::dvec4 *d_min_bounds, *d_max_bounds;
        hipMalloc(&d_min_bounds, sizeof(glm::dvec4));
        hipMalloc(&d_max_bounds, sizeof(glm::dvec4));
        hipMemcpy(d_min_bounds, &h_min_bounds, sizeof(glm::dvec4), hipMemcpyHostToDevice);
        hipMemcpy(d_max_bounds, &h_max_bounds, sizeof(glm::dvec4), hipMemcpyHostToDevice);

        compute_aabb<<<gridSize, blockSize>>>(d_positions, num_nodes, d_min_bounds, d_max_bounds);
        hipMemcpy(&h_min_bounds, d_min_bounds, sizeof(glm::dvec4), hipMemcpyDeviceToHost);
        hipMemcpy(&h_max_bounds, d_max_bounds, sizeof(glm::dvec4), hipMemcpyDeviceToHost);

        hipFree(d_min_bounds);
        hipFree(d_max_bounds);

        // Calculate bin size dynamically
        h_bin_size = (h_max_bounds - h_min_bounds) / double(GRID_SIZE);

        // Step 2: Compute node bin mapping on the host
        int* host_node_bins = new int[num_nodes];
        compute_node_bins(host_positions, host_node_bins, num_nodes, h_min_bounds, h_bin_size);

        // Allocate memory for sorted positions and node bins
        glm::dvec4* sorted_positions = new glm::dvec4[num_nodes];
        int* sorted_node_bins = new int[num_nodes];
        int* original_indices = new int[num_nodes];
        int* sorted_indices = new int[num_nodes];

        for (int i = 0; i < num_nodes; ++i) {
            original_indices[i] = i;
        }

        // Populate bin counts and sort positions
        int bin_counts[GRID_SIZE * GRID_SIZE * GRID_SIZE] = {0};
        for (int i = 0; i < num_nodes; ++i) {
            bin_counts[host_node_bins[i]]++;
        }

        sort_positions_by_bins_with_indices(host_positions, sorted_positions, host_node_bins, 
                                            sorted_node_bins, original_indices, sorted_indices, 
                                            num_nodes, bin_counts);

        // Copy sorted data to device
        hipMemcpy(d_positions, sorted_positions, size, hipMemcpyHostToDevice);
        hipMemcpy(d_node_bins, sorted_node_bins, node_bins_size, hipMemcpyHostToDevice);

        int* d_sorted_indices;
        hipMalloc(&d_sorted_indices, num_nodes * sizeof(int));
        hipMemcpy(d_sorted_indices, sorted_indices, num_nodes * sizeof(int), hipMemcpyHostToDevice);

        // Step 3: Populate bins
        populate_bins<<<gridSize, blockSize>>>(d_positions, d_bins, num_nodes, h_min_bounds, h_bin_size);

        // Step 4: Finalize bins
        finalize_bins<<<(GRID_SIZE * GRID_SIZE * GRID_SIZE + blockSize - 1) / blockSize, blockSize>>>(d_bins, GRID_SIZE * GRID_SIZE * GRID_SIZE);

        // Step 5: Compute repulsion forces
        compute_repulsion_kernel_binned<<<gridSize, blockSize>>>(d_positions, d_velocity_deltas, d_bins, 
                                                                 d_node_bins, d_sorted_indices, num_nodes, 
                                                                 repel_force, h_min_bounds, h_bin_size);

        // Cleanup
        delete[] sorted_positions;
        delete[] sorted_node_bins;
        delete[] original_indices;
        delete[] host_node_bins;
        hipFree(d_bins);
        hipFree(d_node_bins);
        hipFree(d_sorted_indices);
    }

    // Copy results back to host
    hipMemcpy(host_velocity_deltas, d_velocity_deltas, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_positions);
    hipFree(d_velocity_deltas);
}
