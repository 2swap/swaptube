#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include "../DataObjects/PendulumHelpers.cpp"

// Kernel to update pendulum states
__global__ void pendulum_simulation_kernel(
    PendulumState* d_states,
    int n,
    int multiplier,
    double dt
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Compute the next state using the shared RK4 step function
    for(int i = 0; i < multiplier; i++) d_states[idx] = rk4Step(d_states[idx], dt);
}

// Host-exposed function to simulate the pendulum
extern "C" void simulatePendulum(
    PendulumState* states, // Pointer to pendulum states in host memory
    int n,                 // Number of pendulums
    int multiplier,
    double dt              // Time step
) {
    PendulumState* d_states;

    // Allocate memory on the device for pendulum states
    hipMalloc(&d_states, n * sizeof(PendulumState));

    // Copy initial states from host to device
    hipMemcpy(d_states, states, n * sizeof(PendulumState), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    pendulum_simulation_kernel<<<numBlocks, threadsPerBlock>>>(d_states, n, multiplier, dt);

    // Copy updated states back from device to host
    hipMemcpy(states, d_states, n * sizeof(PendulumState), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_states);
}

// Kernel to update pendulum states
__global__ void double_pendulum_simulation_kernel(
    PendulumState* d_states,
    PendulumState* d_pairs,
    double* d_diffs,
    int n,
    int multiplier,
    double dt
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Compute the next state using the shared RK4 step function
    for(int i = 0; i < multiplier; i++) {
        d_states[idx] = rk4Step(d_states[idx], dt);
        d_pairs[idx] = rk4Step(d_pairs[idx], dt);
        double p1_dist = d_states[idx].p1 - d_pairs[idx].p1;
        double p2_dist = d_states[idx].p2 - d_pairs[idx].p2;
        double theta1_dist = d_states[idx].theta1 - d_pairs[idx].theta1;
        double theta2_dist = d_states[idx].theta2 - d_pairs[idx].theta2;
        double distance = sqrt(p1_dist*p1_dist + p2_dist*p2_dist + theta1_dist*theta1_dist + theta2_dist*theta2_dist);
        distance = min(distance, 1.);
        d_diffs[idx] += distance;
    }
}

// Host-exposed function to simulate the pendulum
extern "C" void simulate_pendulum_pair(
    PendulumState* states, // Pointer to pendulum states in host memory
    PendulumState* pairs, // Pointer to pendulum states in host memory
    double* diffs, // Pointer to pendulum states in host memory
    int n,                 // Number of pendulums
    int multiplier,
    double dt              // Time step
) {
    PendulumState* d_states;
    PendulumState* d_pairs;
    double* d_diffs;

    // Allocate memory on the device for pendulum states
    hipMalloc(&d_states, n * sizeof(PendulumState));
    hipMalloc(&d_pairs, n * sizeof(PendulumState));
    hipMalloc(&d_diffs, n * sizeof(double));

    // Copy initial states from host to device
    hipMemcpy(d_states, states, n * sizeof(PendulumState), hipMemcpyHostToDevice);
    hipMemcpy(d_pairs, pairs, n * sizeof(PendulumState), hipMemcpyHostToDevice);
    hipMemcpy(d_diffs, diffs, n * sizeof(double), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    double_pendulum_simulation_kernel<<<numBlocks, threadsPerBlock>>>(d_states, d_pairs, d_diffs, n, multiplier, dt);

    // Copy updated states back from device to host
    hipMemcpy(states, d_states, n * sizeof(PendulumState), hipMemcpyDeviceToHost);
    hipMemcpy(pairs, d_pairs, n * sizeof(PendulumState), hipMemcpyDeviceToHost);
    hipMemcpy(diffs, d_diffs, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_states);
    hipFree(d_pairs);
    hipFree(d_diffs);
}

