#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <glm/glm.hpp>
#include <complex>
#include <hip/hip_complex.h>  // Use hipDoubleComplex for complex numbers in CUDA

const double bailout_radius = 256;
const double bailout_radius_sq = bailout_radius*bailout_radius;

// Function to linearly interpolate between two colors
__device__ unsigned int cuda_color_lerp(unsigned int c1, unsigned int c2, double t) {
    return ((unsigned int)((1 - t) * ((c1 >> 24) & 0xff) + t * ((c2 >> 24) & 0xff)) << 24) |
           ((unsigned int)((1 - t) * ((c1 >> 16) & 0xff) + t * ((c2 >> 16) & 0xff)) << 16) |
           ((unsigned int)((1 - t) * ((c1 >> 8 ) & 0xff) + t * ((c2 >> 8 ) & 0xff)) << 8 ) |
           ((unsigned int)((1 - t) * ( c1        & 0xff) + t * ( c2        & 0xff))      ) ;
}

__device__ hipDoubleComplex cuCpow(hipDoubleComplex base, hipDoubleComplex exponent) {
    double a = hipCreal(base);
    double b = hipCimag(base);
    double c = hipCreal(exponent);
    double d = hipCimag(exponent);
    if (a == 0.0 && b == 0.0)
        return make_hipDoubleComplex(0.0, 0.0);  // Zero raised to positive power is zero
    
    double r = sqrt(a * a + b * b);  // Magnitude of the base
    double theta = atan2(b, a);      // Argument of the base

    double new_r = pow(r, c) * exp(-d * theta);
    double new_theta = c * theta + d * log(r);

    return make_hipDoubleComplex(new_r * cos(new_theta), new_r * sin(new_theta));
}

// Color interpolation function (shared)
__device__ unsigned int get_mandelbrot_color(double iterations, int max_iterations, bool bailed_out, double gradation, double sq_radius, double log_real_part_exp, double breath, unsigned int internal_color) {
    if(!bailed_out) return internal_color;

    if(bailed_out && gradation > 0.01){
        double log_zn = log(sq_radius)/2;
        double nu = log(log_zn / log_real_part_exp) / log_real_part_exp;
        iterations += (1-nu) * gradation; // Do not use gradient for exponential parameterization
    }

    const unsigned int color_palette[] = {
        0xffffffff,
        0xff000088,
        0xff000000,
        0xff000088,
    };
    /*const unsigned int color_palette[] = {
        0xff0e7c4a,
        0xff2258a5,
        0xff002347,
        0xff000000,
    };*/
    /*const unsigned int color_palette[] = {
        0xff5d0e41,
        0xff00224d,
        0xff000000,
    };*/
    const int palette_size = sizeof(color_palette) / sizeof(color_palette[0]);

    iterations = (iterations + 50 - breath) / 5.;
    int idx = floor(iterations);
    double w = iterations - idx;
    idx %= palette_size;
    return cuda_color_lerp(color_palette[idx], color_palette[(idx + 1) % palette_size], w);
}

__device__ void compute_z_x_c(
    int pixel_x, int pixel_y, int width, int height,
    const hipDoubleComplex seed_z, const hipDoubleComplex seed_x, const hipDoubleComplex seed_c,
    const glm::vec3 pixel_parameter_multipliers,
    const hipDoubleComplex zoom,
    hipDoubleComplex& z, hipDoubleComplex& x, hipDoubleComplex& c, double& log_real_part_exp
) {
    // Calculate the complex point based on pixel coordinates
    hipDoubleComplex point = make_hipDoubleComplex(
        4 * ((pixel_x - width / 2.0) / static_cast<float>(height)),
        4 * ((static_cast<float>(pixel_y) / height) - 0.5f)
    );

    point = hipCmul(point, zoom);

    // Compute z, x, and c based on seed values and multipliers
    z = hipCadd(seed_z, hipCmul(make_hipDoubleComplex(pixel_parameter_multipliers.x, 0), point));
    x = hipCadd(seed_x, hipCmul(make_hipDoubleComplex(pixel_parameter_multipliers.y, 0), point));
    c = hipCadd(seed_c, hipCmul(make_hipDoubleComplex(pixel_parameter_multipliers.z, 0), point));
    double rpe = hipCreal(x);
    log_real_part_exp = log(rpe);
}

__device__ int mandelbrot_iterations(
    hipDoubleComplex &z, const hipDoubleComplex &x, const hipDoubleComplex &c,
    int max_iterations, double bailout_radius_sq, double &sq_radius
) {
    int iterations = 0;
    sq_radius = 0;
    
    for (; iterations < max_iterations; iterations++) {
        z = hipCadd(cuCpow(z, x), c);
        double r = hipCreal(z);
        double i = hipCimag(z);
        sq_radius = r * r + i * i;
        if (sq_radius > bailout_radius_sq) {
            return iterations; // Returns immediately if bailout occurs
        }
    }
    
    return max_iterations; // No bailout, maximum iterations reached
}

__device__ int mandelbrot_iterations_2or3(
    hipDoubleComplex &z, int exponent, const hipDoubleComplex &c,
    int max_iterations, double bailout_radius_sq, double &sq_radius
) {
    int iterations = 0;
    sq_radius = 0;

    // Extract real and imaginary parts of z and c
    double zr = hipCreal(z);
    double zi = hipCimag(z);
    double cr = hipCreal(c);
    double ci = hipCimag(c);

    if(exponent == 2){
        for (; iterations < max_iterations; iterations++) {
            double zr_new = zr * zr - zi * zi + cr;  // Real part of z^2 + c
            double zi_new = 2.0 * zr * zi + ci;      // Imaginary part of z^2 + c

            // Update z and square radius for next iteration
            zr = zr_new;
            zi = zi_new;
            sq_radius = zr * zr + zi * zi;

            if (sq_radius > bailout_radius_sq) return iterations;
        }
    } else {
        for (; iterations < max_iterations; iterations++) {
            double zr_new = zr * zr * zr - 3.0 * zr * zi * zi + cr;  // Real part of z^3 + c
            double zi_new = 3.0 * zr * zr * zi - zi * zi * zi + ci;  // Imaginary part of z^3 + c

            // Update z and square radius for next iteration
            zr = zr_new;
            zi = zi_new;
            sq_radius = zr * zr + zi * zi;

            if (sq_radius > bailout_radius_sq) return iterations;
        }
    }

    return max_iterations; // No bailout, maximum iterations reached
}

__global__ void go(
    const int width, const int height,
    const hipDoubleComplex seed_z, const hipDoubleComplex seed_x, const hipDoubleComplex seed_c,
    const glm::vec3 pixel_parameter_multipliers,
    const hipDoubleComplex zoom,
    int max_iterations,
    float gradation,
    float breath,
    unsigned int internal_color,
    unsigned int* colors
) {
    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (pixel_x >= width || pixel_y >= height) return;

    hipDoubleComplex z, x, c; 
    double log_real_part_exp, sq_radius = 0;
    compute_z_x_c(pixel_x, pixel_y, width, height, seed_z, seed_x, seed_c, pixel_parameter_multipliers, zoom, z, x, c, log_real_part_exp);

    // Check if the exponent 'x' is a positive integer
    bool x_is_real = (hipCimag(x) == 0) && (hipCreal(x) > 0) && (hipCreal(x) == (int)hipCreal(x));
    int intx = hipCreal(x);

    int iterations;
    if (x_is_real && (intx == 2 || intx == 3)) {
        iterations = mandelbrot_iterations_2or3(z, intx, c, max_iterations, bailout_radius_sq, sq_radius);
    } else {
        iterations = mandelbrot_iterations(z, x, c, max_iterations, bailout_radius_sq, sq_radius);
    }
    
    bool bailed_out = iterations < max_iterations;

    colors[pixel_y * width + pixel_x] = get_mandelbrot_color(iterations, max_iterations, bailed_out, gradation, sq_radius, log_real_part_exp, breath, internal_color);
}

// Host function to launch the kernel
extern "C" void mandelbrot_render(
    const int width, const int height,
    const std::complex<double> seed_z, const std::complex<double> seed_x, const std::complex<double> seed_c,
    const glm::vec3 pixel_parameter_multipliers,
    const std::complex<double> zoom,
    int max_iterations,  // Pass max_iterations as a parameter
    float gradation,
    float breath,
    unsigned int internal_color,
    unsigned int* colors
) {
    unsigned int* d_colors;

    // Allocate memory on the device for the depth buffer
    hipMalloc(&d_colors, width * height * sizeof(unsigned int));

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);  // 2D block of 16x16 threads
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    go<<<numBlocks, threadsPerBlock>>>(
        width, height,
        make_hipDoubleComplex(seed_z.real(), seed_z.imag()), make_hipDoubleComplex(seed_x.real(), seed_x.imag()), make_hipDoubleComplex(seed_c.real(), seed_c.imag()),
        pixel_parameter_multipliers,
        make_hipDoubleComplex(zoom.real(), zoom.imag()),
        max_iterations, gradation, breath, internal_color, d_colors
    );

    // Copy results back from device to host
    hipMemcpy(colors, d_colors, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_colors);
}
