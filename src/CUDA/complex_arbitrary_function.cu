#include "hip/hip_runtime.h"
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../Host_Device_Shared/helpers.h"
#include "color.cuh"

__device__ thrust::complex<float> complex_sqrt(const thrust::complex<float>& z, float branch_cut) {
    float r = thrust::abs(z);
    float theta = atan2(z.imag(), z.real());
    if (theta < branch_cut) {
        theta += 2 * M_PI; // Adjust angle based on branch cut
    }
    return thrust::polar(sqrt(r), theta / 2);
}

__global__ void render_kernel(
    int* d_pixels,
    glm::vec2 wh,
    float sqrt_coef, float sqrt_branch_cut, float sin_coef, float cos_coef, float exp_coef,
    glm::vec2 lx_ty,
    glm::vec2 rx_by,
    float ab_dilation,
    float dot_radius
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= wh.x || y >= wh.y) return;

    const glm::vec2 point = pixel_to_point(glm::vec2(x,y), lx_ty, rx_by, wh);
    thrust::complex<float> val;
    const thrust::complex<float> sqrt_val = complex_sqrt(val, sqrt_branch_cut);
    const thrust::complex<float> sin_val = thrust::sin(val);
    const thrust::complex<float> cos_val = thrust::cos(val);
    const thrust::complex<float> exp_val = thrust::exp(val);
    const thrust::complex<float> sum = sqrt_coef * sqrt_val + sin_coef * sin_val + cos_coef * cos_val + exp_coef * exp_val;
    const int color = d_complex_to_srgb(sum, ab_dilation, dot_radius);

    d_pixels[y * int(wh.x) + x] = color;
}

extern "C" void color_complex_arbitrary_function(
    unsigned int* h_pixels, // to be overwritten with the result
    int w,
    int h,
    float sqrt_coef, float sqrt_branch_cut, float sin_coef, float cos_coef, float exp_coef,
    float lx, float ty,
    float rx, float by,
    float ab_dilation,
    float dot_radius
) {
    // Allocate device memory for pixels
    int* d_pixels;
    hipMalloc(&d_pixels, w * h * sizeof(int));

    // Define the region in complex plane
    glm::vec2 wh(w, h);
    glm::vec2 lx_ty(lx, ty);
    glm::vec2 rx_by(rx, by);

    // Kernel config
    dim3 blockSize(16, 16);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    render_kernel<<<gridSize, blockSize>>>(d_pixels, wh, sqrt_coef, sqrt_branch_cut, sin_coef, cos_coef, exp_coef, lx_ty, rx_by, ab_dilation, dot_radius);
    hipDeviceSynchronize();

    // Copy pixels back to host
    hipMemcpy(h_pixels, d_pixels, w * h * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_pixels);
}
