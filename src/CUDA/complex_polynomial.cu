#include "hip/hip_runtime.h"
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include "../Host_Device_Shared/helpers.h"
#include "color.cuh"
#include <cmath>
#include <glm/glm.hpp>

__device__ thrust::complex<float> evaluate_polynomial_given_coefficients(const thrust::complex<float>* coefficients, int degree, const thrust::complex<float>& point) {
    thrust::complex<float> result(0.0, 0.0);
    thrust::complex<float> power_of_point(1.0, 0.0);
    for (int i = 0; i <= degree; i++) {
        result += coefficients[i] * power_of_point;
        power_of_point *= point;
    }
    return result;
}

__global__ void render_kernel(
    int* d_pixels,
    const thrust::complex<float>* d_coefficients,
    int degree,
    glm::vec2 wh,
    glm::vec2 lx_ty,
    glm::vec2 rx_by,
    float ab_dilation,
    float dot_radius
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= wh.x || y >= wh.y) return;

    const glm::vec2 point = pixel_to_point(glm::vec2(x,y), lx_ty, rx_by, wh);
    const thrust::complex<float> val = evaluate_polynomial_given_coefficients(d_coefficients, degree, thrust::complex<float>(point.x, point.y));
    const int color = d_complex_to_srgb(val, ab_dilation, dot_radius);

    d_pixels[y * int(wh.x) + x] = color;
}

extern "C" void color_complex_polynomial(
    unsigned int* h_pixels, // to be overwritten with the result
    int w,
    int h,
    const float* h_coefficients_real,
    const float* h_coefficients_imag,
    int degree,
    float lx, float ty,
    float rx, float by,
    float ab_dilation,
    float dot_radius
) {
    // Allocate device memory for pixels
    int* d_pixels;
    hipMalloc(&d_pixels, w * h * sizeof(int));

    // Allocate device memory for coefficients
    thrust::complex<float>* d_coefficients;
    hipMalloc(&d_coefficients, (degree + 1) * sizeof(thrust::complex<float>));

    // Create host array of complex coeffs
    thrust::complex<float>* h_coefficients = new thrust::complex<float>[degree + 1];
    for(int i = 0; i <= degree; i++){
        h_coefficients[i] = thrust::complex<float>(h_coefficients_real[i], h_coefficients_imag[i]);
    }

    // Copy coefficients to device
    hipMemcpy(d_coefficients, h_coefficients, (degree + 1) * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
    delete[] h_coefficients;

    // Define the region in complex plane
    glm::vec2 wh(w, h);
    glm::vec2 lx_ty(lx, ty);
    glm::vec2 rx_by(rx, by);

    // Kernel config
    dim3 blockSize(16, 16);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    render_kernel<<<gridSize, blockSize>>>(d_pixels, d_coefficients, degree, wh, lx_ty, rx_by, ab_dilation, dot_radius);
    hipDeviceSynchronize();

    // Copy pixels back to host
    hipMemcpy(h_pixels, d_pixels, w * h * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_pixels);
    hipFree(d_coefficients);
}
