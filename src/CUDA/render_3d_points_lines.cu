#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <utility>
#include <glm/glm.hpp>
#include <glm/gtc/quaternion.hpp>
#include "../Scenes/Common/ThreeDimensionStructs.cpp"
#include "../misc/cuda_color.cu" // Contains overlay_pixel and set_pixel

__device__ void device_coordinate_to_pixel(
    const glm::vec3& coordinate,
    bool &behind_camera,
    const glm::quat& camera_direction,
    const glm::vec3& camera_pos,
    const glm::quat& conjugate_camera_direction,
    float fov,
    float geom_mean_size,
    int width,
    int height,
    float& outx,
    float& outy)
{
    glm::vec3 rotated = camera_direction * (coordinate - camera_pos) * conjugate_camera_direction;
    if (rotated.z <= 0) { behind_camera = true; return; }
    float scale = (geom_mean_size * fov) / rotated.z;
    outx = scale * rotated.x + width * 0.5f;
    outy = scale * rotated.y + height * 0.5f;
}

__device__ void device_fill_circle(float cx, float cy, float r, int col, unsigned int* pixels, int width, int height, float opa=1.0f) {
    for (float dx = -r; dx < r; dx++) {
        float sdx = square(dx);
        for (float dy = -r; dy < r; dy++) {
            if (sdx + square(dy) < r*r)
                //overlay_pixel(cx + dx, cy + dy, col, opa, pixels, width, height);
                    set_pixel(cx + dx, cy + dy, col     , pixels, width, height);
        }
    }
}

__device__ __forceinline__ void bresenham(int x1, int y1, int x2, int y2, int col, float opacity, int thickness, unsigned int* pixels, int width, int height) {
    int dx = abs(x2 - x1), dy = abs(y2 - y1);
    if (dx > 10000 || dy > 10000) return;
    int sx = (x1 < x2) ? 1 : -1;
    int sy = (y1 < y2) ? 1 : -1;
    int err = dx - dy;

    while (true) {
        set_pixel(x1, y1, col,          pixels, width, height);
        //overlay_pixel(x1, y1, col, opacity, pixels, width, height);
        for (int i = 1; i < thickness; i++) {
            set_pixel(x1 + i, y1, col, pixels, width, height);
            set_pixel(x1 - i, y1, col, pixels, width, height);
            set_pixel(x1, y1 + i, col, pixels, width, height);
            set_pixel(x1, y1 - i, col, pixels, width, height);
            //overlay_pixel(x1 + i, y1, col, opacity, pixels, width, height);
            //overlay_pixel(x1 - i, y1, col, opacity, pixels, width, height);
            //overlay_pixel(x1, y1 + i, col, opacity, pixels, width, height);
            //overlay_pixel(x1, y1 - i, col, opacity, pixels, width, height);
        }
        if (x1 == x2 && y1 == y2) break;
        int e2 = 2 * err;
        if (e2 > -dy) { err -= dy; x1 += sx; }
        if (e2 <  dx) { err += dx; y1 += sy; }
    }
}

__global__ void render_points_kernel(
    unsigned int* pixels, int width, int height,
    float geom_mean_size, float points_opacity, float points_radius_multiplier,
    Point* points, int num_points,
    glm::quat camera_direction, glm::vec3 camera_pos, glm::quat conjugate_camera_direction, float fov)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= num_points) return;
    Point p = points[idx];
    if (p.opacity == 0) return;
    bool behind_camera = false;
    float px, py;
    device_coordinate_to_pixel(
        p.center, behind_camera,
        camera_direction, camera_pos, conjugate_camera_direction, fov,
        geom_mean_size, width, height, px, py);
    if (behind_camera) return;
    float dot_size = p.size * points_radius_multiplier * geom_mean_size / 400.0f;
    device_fill_circle(px, py, dot_size, p.color, pixels, width, height, points_opacity * p.opacity);
}

__global__ void render_lines_kernel(
    unsigned int* pixels, int width, int height,
    float geom_mean_size, int thickness, float lines_opacity,
    Line* lines, int num_lines,
    glm::quat camera_direction, glm::vec3 camera_pos, glm::quat conjugate_camera_direction, float fov)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= num_lines) return;
    Line ln = lines[idx];
    if (ln.opacity == 0) return;
    bool behind_camera1 = false, behind_camera2 = false;
    float p1x, p1y, p2x, p2y;
    device_coordinate_to_pixel(
        ln.start, behind_camera1,
        camera_direction, camera_pos, conjugate_camera_direction, fov,
        geom_mean_size, width, height, p1x, p1y);
    device_coordinate_to_pixel(
        ln.end,   behind_camera2,
        camera_direction, camera_pos, conjugate_camera_direction, fov,
        geom_mean_size, width, height, p2x, p2y);
    if (behind_camera1 || behind_camera2) return;
    bresenham(
        p1x, p1y, p2x, p2y,
        ln.color, lines_opacity * ln.opacity, thickness,
        pixels, width, height);
}

extern "C" void render_points_on_gpu(
    unsigned int* h_pixels, int width, int height,
    float geom_mean_size, float points_opacity, float points_radius_multiplier,
    Point* h_points, int num_points,
    glm::quat camera_direction, glm::vec3 camera_pos, glm::quat conjugate_camera_direction, float fov)
{
    unsigned int* d_pixels = nullptr;
    Point*        d_points = nullptr;
    size_t pix_sz = width * height * sizeof(unsigned int);
    size_t pt_sz  = num_points * sizeof(Point);

    hipMalloc((void**)&d_pixels, pix_sz);
    hipMemcpy(d_pixels, h_pixels, pix_sz, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_points, pt_sz);
    hipMemcpy(d_points, h_points, pt_sz, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_points + blockSize - 1) / blockSize;
    render_points_kernel<<<numBlocks, blockSize>>>(
        d_pixels, width, height,
        geom_mean_size, points_opacity, points_radius_multiplier,
        d_points, num_points,
        camera_direction, camera_pos, conjugate_camera_direction, fov);
    hipDeviceSynchronize();

    hipMemcpy(h_pixels, d_pixels, pix_sz, hipMemcpyDeviceToHost);
    hipFree(d_pixels);
    hipFree(d_points);
}

extern "C" void render_lines_on_gpu(
    unsigned int* h_pixels, int width, int height,
    float geom_mean_size, int thickness, float lines_opacity,
    Line* h_lines, int num_lines,
    glm::quat camera_direction, glm::vec3 camera_pos, glm::quat conjugate_camera_direction, float fov)
{
    unsigned int* d_pixels = nullptr;
    Line*         d_lines  = nullptr;
    size_t pix_sz = width * height * sizeof(unsigned int);
    size_t ln_sz  = num_lines * sizeof(Line);

    hipMalloc((void**)&d_pixels, pix_sz);
    hipMemcpy(d_pixels, h_pixels, pix_sz, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_lines, ln_sz);
    hipMemcpy(d_lines, h_lines, ln_sz, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_lines + blockSize - 1) / blockSize;
    render_lines_kernel<<<numBlocks, blockSize>>>(
        d_pixels, width, height,
        geom_mean_size, thickness, lines_opacity,
        d_lines, num_lines,
        camera_direction, camera_pos, conjugate_camera_direction, fov);
    hipDeviceSynchronize();

    hipMemcpy(h_pixels, d_pixels, pix_sz, hipMemcpyDeviceToHost);
    hipFree(d_pixels);
    hipFree(d_lines);
}
