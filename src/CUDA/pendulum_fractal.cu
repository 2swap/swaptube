#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include "../Host_Device_Shared/PendulumHelpers.cpp"

// Kernel to update pendulum states
__global__ void pendulum_simulation_kernel(
    Cuda::PendulumState* d_states,
    int n,
    int multiplier,
    Cuda::pendulum_type dt
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Compute the next state using the shared RK4 step function
    for(int i = 0; i < multiplier; i++) d_states[idx] = rk4Step(d_states[idx], dt);
}

// Host-exposed function to simulate the pendulum
extern "C" void simulatePendulum(
    Cuda::PendulumState* states, // Pointer to pendulum states in host memory
    int n,                 // Number of pendulums
    int multiplier,
    Cuda::pendulum_type dt              // Time step
) {
    Cuda::PendulumState* d_states;

    // Allocate memory on the device for pendulum states
    hipMalloc(&d_states, n * sizeof(Cuda::PendulumState));

    // Copy initial states from host to device
    hipMemcpy(d_states, states, n * sizeof(Cuda::PendulumState), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    pendulum_simulation_kernel<<<numBlocks, threadsPerBlock>>>(d_states, n, multiplier, dt);

    // Copy updated states back from device to host
    hipMemcpy(states, d_states, n * sizeof(Cuda::PendulumState), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_states);
}

// Kernel to update pendulum states
__global__ void double_pendulum_simulation_kernel(
    Cuda::PendulumState* d_states,
    Cuda::PendulumState* d_pairs,
    Cuda::pendulum_type* d_diffs,
    int n,
    int multiplier,
    Cuda::pendulum_type dt
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Compute the next state using the shared RK4 step function
    for(int i = 0; i < multiplier; i++) {
        d_states[idx] = rk4Step(d_states[idx], dt);
        d_pairs[idx] = rk4Step(d_pairs[idx], dt);
        Cuda::pendulum_type p1_dist = d_states[idx].p1 - d_pairs[idx].p1;
        Cuda::pendulum_type p2_dist = d_states[idx].p2 - d_pairs[idx].p2;
        Cuda::pendulum_type theta1_dist = d_states[idx].theta1 - d_pairs[idx].theta1;
        Cuda::pendulum_type theta2_dist = d_states[idx].theta2 - d_pairs[idx].theta2;
        Cuda::pendulum_type distance = sqrt(p1_dist*p1_dist + p2_dist*p2_dist + theta1_dist*theta1_dist + theta2_dist*theta2_dist);
        distance = min(distance, 1.f);
        d_diffs[idx] += distance;
    }
}

// Host-exposed function to simulate the pendulum
extern "C" void simulate_pendulum_pair(
    Cuda::PendulumState* states, // Pointer to pendulum states in host memory
    Cuda::PendulumState* pairs, // Pointer to pendulum states in host memory
    Cuda::pendulum_type* diffs, // Pointer to pendulum states in host memory
    int n,                 // Number of pendulums
    int multiplier,
    Cuda::pendulum_type dt              // Time step
) {
    Cuda::PendulumState* d_states;
    Cuda::PendulumState* d_pairs;
    Cuda::pendulum_type* d_diffs;

    // Allocate memory on the device for pendulum states
    hipMalloc(&d_states, n * sizeof(Cuda::PendulumState));
    hipMalloc(&d_pairs , n * sizeof(Cuda::PendulumState));
    hipMalloc(&d_diffs , n * sizeof(Cuda::pendulum_type));

    // Copy initial states from host to device
    hipMemcpy(d_states, states, n * sizeof(Cuda::PendulumState), hipMemcpyHostToDevice);
    hipMemcpy(d_pairs , pairs , n * sizeof(Cuda::PendulumState), hipMemcpyHostToDevice);
    hipMemcpy(d_diffs , diffs , n * sizeof(Cuda::pendulum_type), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    double_pendulum_simulation_kernel<<<numBlocks, threadsPerBlock>>>(d_states, d_pairs, d_diffs, n, multiplier, dt);

    // Copy updated states back from device to host
    hipMemcpy(states, d_states, n * sizeof(Cuda::PendulumState), hipMemcpyDeviceToHost);
    hipMemcpy(pairs , d_pairs , n * sizeof(Cuda::PendulumState), hipMemcpyDeviceToHost);
    hipMemcpy(diffs , d_diffs , n * sizeof(Cuda::pendulum_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_states);
    hipFree(d_pairs);
    hipFree(d_diffs);
}

